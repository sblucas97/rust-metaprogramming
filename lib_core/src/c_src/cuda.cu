#include<stdio.h>
#include<hip/hip_runtime.h>

#define ROWS 10
#define COLS 10

void checkCudaError(hipError_t err, const char *msg) {
	if (err != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s: %s \n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	fprintf(stderr, "EVERYTHING OK\n");
}


__global__ void matrix_add_kernel(const float *a, const float *b, float *result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < ROWS * COLS) {
        result[idx] = a[idx] + b[idx];
    }
}

extern "C" void allocate_gpu_mem(float **a_d) {
	size_t size = ROWS * COLS * sizeof(float);
	hipError_t err = hipMalloc((void**)a_d, size);
	checkCudaError(err, "Failed to cuda malloc");
}

extern "C" void copy_to_gpu(float *a_d, const float *a_h) {
	size_t size = ROWS * COLS * sizeof(float);
	hipError_t err = hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
	checkCudaError(err, "Failed to allocate device memory");
}

extern "C" void copy_from_gpu(float *result_h, float *result_d) {
	size_t size = ROWS * COLS * sizeof(float);
	hipError_t err = hipMemcpy(result_h, result_d, size, hipMemcpyDeviceToHost);
	checkCudaError(err, "Failed to copy result to device memory");
}

extern "C" void launch_kernel(float *a_d, float *b_d, float *result_d) {
    int total = ROWS * COLS;
    int block_size = 256;
    int grid_size = (total + block_size - 1) / block_size;

    matrix_add_kernel<<<grid_size, block_size>>>(a_d, b_d, result_d);
	hipError_t err = hipGetLastError();
    checkCudaError(err, "Kernel launch failed");
    hipDeviceSynchronize(); // Wait for kernel to finish
}

extern "C" void free_gpu_mem(float *data_device) {
	hipFree(data_device);
}